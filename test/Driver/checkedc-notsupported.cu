
#include <hip/hip_runtime.h>
// Checked C extension is not supported for CUDA.   Make sure driver
// rejects the flag.
//
// RUN: %clang -fcheckedc-extension -nocudalib -nocudainc -fsyntax-only -c %s 2>&1 | FileCheck %s
// CHECK: warning: Checked C extension not supported with 'CUDA'; ignoring '-fcheckedc-extension'
//
// Have clang compile this file as a C file.
// RUN: %clang -c -fcheckedc-extension -x c %s
//
// Have clang-cl compile this file as a C file.
// RUN: %clang_cl -c -Xclang -fcheckedc-extension /TC %s

void f() {}


